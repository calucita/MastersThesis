#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
//#include <immintrin.h>
#include <unistd.h>
#include "cmdline.h"
#include "timeit.h"
//#include "helper_avx_cpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define SIZEXBLOCK 32 
#define SIZEYBLOCK 32


#define MAX_ITERS 100
#define UNROLL 8
#define GiB (1024*1024*1024)
#define FLOPS_PER_ELM 1
#define DEBUG 0

__device__ float gather(float * __restrict__ A, int idx)
{
    return A[idx];
}
/*
__global__ void scalar(float * __restrict__ A, const long N, const long M)
{
    int *__restrict B = (int *)A; // Use first half for sequential indices.
    float sum;
    for (long j = 0; j < M; j++) { 
		sum = 0.0f;
		for (long i = 0; i < N/2; i++) {
			sum += gather(A, B[i]);
		}
        if (N > 1000000000) A[0] = sum; // Avoid compiler's dead code detection.
    }
    if (DEBUG) printf("scalar sum=%f\n", sum);
}
*/
__global__ void simd(float * __restrict__ A, const long N, const long M)
{
    int j=blockIdx.x*blockDim.x+threadIdx.x;
    int *__restrict B = (int *)A; // Use first half for sequential indices.
    float sum;
	sum = 0.0f;
	for (long i = 0; i < M; i++) {
		sum += gather(A, B[j]);
	}
        if (N > 1000000000) A[0] = sum; // Avoid compiler's dead code detection.
}

__global__ void cuda32(float * __restrict__ A, const long N, const long M)
{
    if (threadIdx.x==0){
    
    int j=blockIdx.x;
    int *__restrict B = (int *)A; // Use first half for sequential indices.
    float sum;
	sum = 0.0f;
	for (long i = 0; i < M; i++) {
		sum += gather(A, B[j]);
	}
        if (N > 1000000000) A[0] = sum; // Avoid compiler's dead code detection.
    }
}

void init(float * __restrict__ A, const long N, const long M)
{
    int * __restrict__ B1 = (int *) A;
    float * __restrict__ B2 = (float *) A + N/2;
    #pragma novector
    for (long i = 0; i < N/2; i++) {
        B1[i] = i+N/2;        // Sequential indexes in first half.
        B2[i] = rand()%(N/2); // Random indexes in second half.
    }
}


// Init memory - first touch - second half to random indices within [0;N/2-1].

void callCuda(float * __restrict__ A, const long N, const long M){
 
    int sizeXBlock = SIZEXBLOCK;
    int sizeXGrid  = (N/2+sizeXBlock-1)/sizeXBlock;

    simd<<< sizeXGrid, sizeXBlock >>>(A, N, M); 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

}

void callCuda32(float * __restrict__ A, const long N, const long M){
 
    int sizeXBlock = SIZEXBLOCK;
    int sizeXGrid  = N/2;

    cuda32<<< sizeXGrid, sizeXBlock >>>(A, N, M); 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

}

int main(int argc, char *argv[])
{
    // Get command line arguments.
    long N, M;
    // Allocate array aligned at page size (4KB).
    cmdline_Bytes_M(argc, argv, 245760, 200000, &N, &M);
	long pagesize = sysconf(_SC_PAGESIZE);
    long size = N * sizeof(float);
//    float *A = (float*) _mm_malloc(size, pagesize);
    float *A;
    checkCudaErrors(hipMalloc((void**)&A,N*sizeof(float)));
 
    float *B = (float*) malloc(size);
    double ti = timeit_mt(init, B, N, M, 1, 1);
    checkCudaErrors(hipMemcpy(A,B, N*sizeof(float),hipMemcpyHostToDevice));
    printf("Init runtime [ms]:              %7.2lf\n", 1000 * ti);


    // Init memory - first touch - is necessary to get consistent results.
//    double ti = timeit_mt(init, A, N, M, 1, 1);
//    printf("Init runtime [ms]:              %7.2lf\n", 1000 * ti);
    printf("----------------------------------------\n");

    // Time kernels for min 1 secs, MAX_ITERS iters
    double t1 = timeit_mt(callCuda, A, N, M, 1, MAX_ITERS);
    printf("Cuda [ms]:                      %7.2lf\n", 1000 * t1);

    ti = timeit_mt(init, B, N, M, 1, 1);
    checkCudaErrors(hipMemset(A,0,N*sizeof(float)));
    double t2 = timeit_mt(callCuda32, A, N, M, 1, MAX_ITERS);
    printf("Cuda (32 elements) [ms]:        %7.2lf\n", 1000 * t2);
//    checkCudaErrors(hipMemcpy(A,Ac, N*sizeof(float),hipMemcpyDeviceToHost));
    printf("----------------------------------------\n");

    // Performance.
    printf("SIMD performance [Gflops]:      %7.2lf\n", M * N * FLOPS_PER_ELM / t1 / GiB);
    printf("----------------------------------------\n");

    // Bandwidth.
    printf("SIMD bandwidth [GB/s]:          %7.2lf\n", M * size / t1 / GiB);
    // Speed-up.
    printf("Speed-up Cuda32/Cuda:           %7.2lf\n", t2 / t1);
    printf("========================================\n");

    // Clean up.
    checkCudaErrors(hipFree(A));
}
