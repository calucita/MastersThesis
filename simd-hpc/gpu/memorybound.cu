#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
//#include <immintrin.h>
#include <unistd.h>
#include "cmdline.h"
#include "timeit.h"
//#include "helper_avx_cpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define SIZEXBLOCK 32 
#define SIZEYBLOCK 32

#define MAX_ITERS 100
#define UNROLL_SCALAR 8
#define UNROLL_SIMD 4
#define GiB (1024*1024*1024)
#define FLOPS_PER_ELM (2)
#define DEBUG 0

void scalar(float * __restrict__ A, const long N, const long M)
{
    float * __restrict__ B = A + N/2; // B is second half of A.
    float sum;
    #pragma nounroll_and_jam // To avoid loop interchange.
    #pragma novector
    for (long j = 0; j < M; j++) { 
        sum = 0.0f;
        #pragma novector
        //#pragma unroll(UNROLL_SCALAR)
        for (long i = 0; i < N/2; i++) { 
            sum += A[i]*B[i];
        }
        if (N > 1000000000) A[0] = sum; // Avoid compiler's dead code detection.
    }
    if (DEBUG) printf("scalar sum=%f\n", sum);
}

__global__ void  cuda(float * __restrict__ A, const long N, const long M)
{
    int j=blockIdx.x*blockDim.x+threadIdx.x;
    float * __restrict__ B = A + N/2; // B is second half of A.
    float sum;
   // #pragma nounroll_and_jam // To avoid loop interchange.
   // #pragma novector
    //for (long j = 0; j < M; j++) { 
        sum = 0.0f;
       // #pragma novector
        //#pragma unroll(UNROLL_SCALAR)
        for (long i = 0; i < M; i++) { 
            sum += A[j]*B[j];
        }
        if (N > 1000000000) A[0] = sum; // Avoid compiler's dead code detection.
   // }

}


void callCuda(float * __restrict__ A, const long N, const long M){
 
    int sizeXBlock = SIZEXBLOCK;
    int sizeXGrid  = (N/2+sizeXBlock-1)/sizeXBlock;

    cuda<<< sizeXGrid, sizeXBlock >>>(A, N, M); 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

}

__global__ void  cuda32(float * __restrict__ A, const long N, const long M)
{
    int j=blockIdx.x;
    if (threadIdx.x==0){
    float * __restrict__ B = A + N/2; // B is second half of A.
    float sum;
   // #pragma nounroll_and_jam // To avoid loop interchange.
   // #pragma novector
    //for (long j = 0; j < M; j++) { 
        sum = 0.0f;
       // #pragma novector
        //#pragma unroll(UNROLL_SCALAR)

        for (long i = 0; i < M; i++) { 
            sum += A[j]*B[j];
        }
        if (N > 1000000000) A[0] = sum; // Avoid compiler's dead code detection.
   // }
      }


}


void callCuda32(float * __restrict__ A, const long N, const long M){
 
    int sizeXBlock = SIZEXBLOCK;
    int sizeXGrid  = (N/2);

    cuda32<<< sizeXGrid, sizeXBlock >>>(A, N, M); 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

}


// Init memory - first touch - second half to random indices within [0;N/2-1].
void init(float * __restrict__ A, const long N, const long M)
{
    float * __restrict__ B = (float *) A;
    #pragma novector
    for (long i = 0; i < N; i++) {
        B[i] = (float)i/N; // Floating point values [0;1[.
    }
}

int main(int argc, char *argv[])
{
    printf("=======================================\n");
    hipSetDevice(2);
 
    // Get command line arguments.
    long N, M;
    cmdline_Bytes_M(argc, argv, 15360, 10000, &N, &M);

    // Allocate array aligned at page size (4KB).
    long pagesize = sysconf(_SC_PAGESIZE);
    long size = N * sizeof(float);
    float *A = (float*) malloc(size);
    
    // Allocate mem in the device
    float *Ac;
    checkCudaErrors(hipMalloc((void**)&Ac,N*sizeof(float)));
    printf("---------------------------------------\n");
 
    // Init memory - first touch - is necessary to get consistent results.
    double ti = timeit_mt(init, A, N, M, 1, 1);
    checkCudaErrors(hipMemset(Ac,0,N*sizeof(float)));
    printf("Init runtime [ms]:              %7.2lf\n", 1000 * ti);
    printf("----------------------------------------\n");

    // Time kernels for min 1 secs, MAX_ITERS iters
    timeit_mt(init, A, N, M, 1, 1);
    double t0 = timeit_mt(scalar, A, N, M, 1, MAX_ITERS);
    printf("Scalar runtime [ms]:            %7.2lf\n", 1000 * t0);
   // if (DEBUG) debug(A, N, M);

    checkCudaErrors(hipMemset(Ac,0,N*sizeof(float)));
    double t1 = timeit_mt(callCuda, Ac, N, M, 1, MAX_ITERS);
    printf("Cuda [ms]:                      %7.2lf\n", 1000 * t1);
    checkCudaErrors(hipMemcpy(A,Ac, N*sizeof(float),hipMemcpyDeviceToHost));
   // if (DEBUG) debug(A, N, M);

    checkCudaErrors(hipMemset(Ac,0,N*sizeof(float)));
    double t2 = timeit_mt(callCuda32, Ac, N, M, 1, MAX_ITERS);
    printf("Cuda (32 elements) [ms]:        %7.2lf\n", 1000 * t2);
    checkCudaErrors(hipMemcpy(A,Ac, N*sizeof(float),hipMemcpyDeviceToHost));
   // if (DEBUG) debug(A, N, M);

   printf("----------------------------------------\n");

    // Performance.
    printf("Scalar performance [Gflops]:    %7.2lf\n", M * N * FLOPS_PER_ELM / t0 / GiB);
    printf("Cuda [Gflops]:                  %7.2lf\n", M * N * FLOPS_PER_ELM / t1 / GiB);
    printf("Cuda (32E) [Gflops]:            %7.2lf\n", M * N * FLOPS_PER_ELM / t2 / GiB);
    printf("----------------------------------------\n");

    // Bandwidth.
    printf("Scalar bandwidth [GB/s]:        %7.2lf\n", size / t0 / GiB);
    printf("Cuda bandwidth [GB/s]:          %7.2lf\n", size / t1 / GiB);
    printf("Cuda32 bandwidth [GB/s]:        %7.2lf\n", size / t2 / GiB);
    printf("----------------------------------------\n");

    // Speed-up.
    printf("Speed-up Cpu/Cuda:              %7.2lf\n", t0 / t1);
    printf("Speed-up Cuda32/Cuda:           %7.2lf\n", t2 / t1);
    printf("========================================\n");

    // Clean up.
    free(A);
    checkCudaErrors(hipFree(Ac));
}
