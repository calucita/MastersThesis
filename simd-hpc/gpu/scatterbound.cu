#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <math.h>
//#include <immintrin.h>
#include <unistd.h>
#include "cmdline.h"
#include "timeit.h"
//#include "helper_avx_cpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define SIZEXBLOCK 32 


#define MAX_ITERS 100
#define UNROLL 8
#define GiB (1024*1024*1024)
#define FLOPS_PER_ELM 1
#define DEBUG 0
__global__ void simd(float * __restrict__ A, const long N, const long M)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int *__restrict__ B = (int *)A + N/2; // Use second half as random indices.

    for (long j = 0; j < M; j++) { 
	float s = A[0];
	A[B[i]] = s;
    }
}
/*
void scalar(float * __restrict__ A, const long N, const long M)
{
    int * __restrict__ B = (int *)A + N/2; // Use second half as random indices.

    #pragma nounroll_and_jam // To avoid loop interchange.
    #pragma novector
    for (long j = 0; j < M; j++) { 
		float s = A[0];
        #pragma novector
		for (long i = 0; i < N/2; i++) {
            __assume_aligned(A, 64);
            __assume_aligned(B, 64);
			A[B[i]] = s;
		}
    }
}
*/
__global__ void cuda32(float * __restrict__ A, const long N, const long M)
{
    if (threadIdx.x ==0){
    int i=blockIdx.x;
    int *__restrict__ B = (int *)A + N/2; // Use second half as random indices.

    for (long j = 0; j < M; j++) { 
	float s = A[0];
	A[B[i]] = s;
    }
    }
}
// Init memory - first touch - second half to random indices within [0;N/2-1].
void init(float * __restrict__ A, const long N, const long M)
{
    int * __restrict__ B1 = (int *) A;
    int * __restrict__ B2 = (int *) A + N/2;
    #pragma novector
    for (long i = 0; i < N/2; i++) {
        B1[i] = i;            // Sequential indexes in first half.
        B2[i] = rand()%(N/2); // Random indexes in second half.
    }
}
void call32(float * __restrict__ A, const long N, const long M){
 
    int sizeXBlock = 32;
    int sizeXGrid  = N/2;

    cuda32<<< sizeXGrid, sizeXBlock >>>(A, N, M); 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

}
void callCuda(float * __restrict__ A, const long N, const long M){
 
    int sizeXBlock = SIZEXBLOCK;
    int sizeXGrid  = (N/2+sizeXBlock-1)/sizeXBlock;

    simd<<< sizeXGrid, sizeXBlock >>>(A, N, M); 
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

}

int main(int argc, char *argv[])
{
    // Get command line arguments.
    long N, M;
    cmdline_Bytes_M(argc, argv, 245760, 200000, &N, &M);
    float *A;
    checkCudaErrors(hipMalloc((void**)&A,N*sizeof(float)));
 
    // Allocate array aligned at page size (4KB).
	long pagesize = sysconf(_SC_PAGESIZE);
//    float *A = (float*) _mm_malloc(N * sizeof(float), pagesize);

     float *B = (float*) malloc(N*sizeof(float));
    double ti = timeit(init, B, N, M, 1, 1);
    checkCudaErrors(hipMemcpy(A,B, N*sizeof(float),hipMemcpyHostToDevice));
    printf("Init runtime [ms]:              %7.2lf\n", 1000 * ti);

   // Init memory - first touch - is necessary to get consistent results.
    // Warm up for min 1 secs, max 1000 iters
    printf("----------------------------------------\n");
    // Time kernels for min 1 secs, max 1000 iters
    double t1 = timeit(callCuda, A, N, M, 1, 1000);
    printf("SIMD runtime [ms]:              %7.2lf\n", 1000 * t1);
    timeit(init, B, N, M, 1, 1);
    checkCudaErrors(hipMemcpy(A,B, N*sizeof(float),hipMemcpyHostToDevice));
    double t2 = timeit(call32, A, N, M, 1, 1000);
    printf("32 runtime [ms]:              %7.2lf\n", 1000 * t2);
    printf("----------------------------------------\n");
    // Performance.
    printf("SIMD performance [Gflops]:      %7.2lf\n", M * N * FLOPS_PER_ELM / t1 / GiB);
    printf("32 performance [Gflops]:      %7.2lf\n", M * N * FLOPS_PER_ELM / t2 / GiB);
    printf("----------------------------------------\n");

    // Bandwidth.
    printf("SIMD bandwidth [GB/s]:          %7.2lf\n", M * sizeof(float)*N / t1 / GiB);
    printf("SIMD bandwidth [GB/s]:          %7.2lf\n", M * sizeof(float)*N / t2 / GiB);
 
    printf("----------------------------------------\n");
    printf("Speed-up Cuda32/Cuda:           %7.2lf\n", t2 / t1);
    printf("=================================\n");

    // Clean up.
    checkCudaErrors(hipFree(A));
}
